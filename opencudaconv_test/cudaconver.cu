#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include "cudaconver.h"

__global__ void rgb2rgbaKernel(const uchar3* src, uchar4* dst, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * width + x;
        uchar3 rgb = src[index];
        dst[index] = make_uchar4(rgb.x, rgb.y, rgb.z, 255);
    }
}

void kernle_convertRGBtoRGBA(const cv::Mat& rgbImage, cv::Mat& rgbaImage){
    // 获取图像尺寸
    int width = rgbImage.cols;
    int height = rgbImage.rows;

    // 计算数据大小
    size_t size = width * height * sizeof(uchar3);

    // 分配CUDA内存
    uchar3* d_src;
    hipMalloc((void**)&d_src, size);

    // 将RGB图像数据复制到CUDA内存
    hipMemcpy(d_src, rgbImage.ptr(), size, hipMemcpyHostToDevice);

    // 分配输出内存
    uchar4* d_dst;
    hipMalloc((void**)&d_dst, size);

    // 定义线程块和网格的大小
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // 调用CUDA核函数进行转换
    rgb2rgbaKernel<<<numBlocks, threadsPerBlock>>>(d_src, d_dst, width, height);

    // 将结果复制回主机内存
    hipMemcpy(rgbaImage.ptr(), d_dst, size, hipMemcpyDeviceToHost);

    // 释放CUDA内存
    hipFree(d_src);
    hipFree(d_dst);
}
